#include "hip/hip_runtime.h"
#include "kernels.h"
#include "common.h"

template <typename T> __global__ 
void vector_square(const T *A_d, T *C_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
}

template <typename T> __global__ 
void silly_lots_of_calcs(const T *A_d, T *C_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] + A_d[i] * A_d[i] - 1.0/(A_d[i]*A_d[i]*A_d[i]) + sin(A_d[i]) + cos(A_d[i])*cos(A_d[i]);
        C_d[i] = sin(C_d[i]) * cos(C_d[i]) + log(std::abs(C_d[i]))*exp(-std::abs(A_d[i]));
        C_d[i] = 1.0/(sin(C_d[i]) * cos(C_d[i]) + log(std::abs(C_d[i]))*exp(-std::abs(A_d[i])));
        for (size_t j=0;j<1000;j++) C_d[i] += C_d[i] = 1.0/(sin(C_d[i]) * cos(C_d[i]) + log(std::abs(C_d[i]))*exp(-std::abs(A_d[i])));
        ;
    }
}

// void silly_template_instantsiation()
// {
//     int ix,iy;
//     float fx,fy;
//     double dx,dy;
//     size_t N;
//     size_t blocksize = 256;
//     size_t threadsperblock = 1024;
//     pu_gpuLaunchKernel(vector_square, 
//         dim3(blocksize), dim3(threadsperblock), 
//         0, 0,
//         &ix, &iy, N);
//     pu_gpuLaunchKernel(vector_square, 
//         dim3(blocksize), dim3(threadsperblock), 
//         0, 0,
//         &fx, &fy, N);
//     pu_gpuLaunchKernel(vector_square, 
//         dim3(blocksize), dim3(threadsperblock), 
//         0, 0,
//         &dx, &dy, N);
// }

void compute_kernel1(size_t N, 
    std::vector<int*> &x_int_gpu, 
    std::vector<int*> &y_int_gpu, 
    std::vector<float*> &x_float_gpu, 
    std::vector<float*> &y_float_gpu, 
    std::vector<double*> &x_double_gpu, 
    std::vector<double*> &y_double_gpu,
    int Niter,
    size_t blocksize,
    size_t threadsperblock
    ) 
{
    int nDevices;
    size_t dynsharedsize = 0;
    pu_gpuStream_t stream = 0;
    pu_gpuErrorCheck(pu_gpuGetDeviceCount(&nDevices));
    Log()<<" Computing kernels ... "<<std::endl;
    for (auto idev=0;idev<nDevices;idev++) {
        pu_gpuErrorCheck(pu_gpuSetDevice(idev));
        auto time_kernel = NewTimer();
        for (auto i=0; i<Niter;i++) {
#ifdef KERNEL2
            pu_gpuLaunchKernel(silly_lots_of_calcs, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_int_gpu[idev], y_int_gpu[idev], N);
#ifdef GPU_DEBUG
            pu_gpuCheckLastKernel();
#endif
            pu_gpuLaunchKernel(silly_lots_of_calcs, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_float_gpu[idev], y_float_gpu[idev], N);
#ifdef GPU_DEBUG
            pu_gpuCheckLastKernel();
#endif
            pu_gpuLaunchKernel(silly_lots_of_calcs, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_double_gpu[idev], y_double_gpu[idev], N);
#ifdef GPU_DEBUG
            pu_gpuCheckLastKernel();
#endif
#else 
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_int_gpu[idev], y_int_gpu[idev], N);
#ifdef GPU_DEBUG
            pu_gpuCheckLastKernel();
#endif
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_float_gpu[idev], y_float_gpu[idev], N);
#ifdef GPU_DEBUG
                pu_gpuCheckLastKernel();
#endif
                pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_double_gpu[idev], y_double_gpu[idev], N);
#ifdef GPU_DEBUG
                pu_gpuCheckLastKernel();
#endif
#endif
        }
        LogTimeTakenOnDevice(time_kernel);
    }
}

// __global__ void vector_square(const int *, int *, size_t);
// __global__ void vector_square(const float *, float *, size_t);
// __global__ void vector_square(const double *, double *, size_t);
