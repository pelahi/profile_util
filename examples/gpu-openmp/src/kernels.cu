#include "hip/hip_runtime.h"
#include "kernels.h"
#include "common.h"

template <typename T> __global__ 
void vector_square(const T *A_d, T *C_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
}

void silly_template_instansiation()
{
    int ix,iy;
    float fx,fy;
    double dx,dy;
    size_t N;
    size_t blocksize = 256;
    size_t threadsperblock = 1024;
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &ix, &iy, N);
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &fx, &fy, N);
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &dx, &dy, N);
}

void compute_kernel1(size_t N, 
    std::vector<int*> &x_int_gpu, 
    std::vector<int*> &y_int_gpu, 
    std::vector<float*> &x_float_gpu, 
    std::vector<float*> &y_float_gpu, 
    std::vector<double*> &x_double_gpu, 
    std::vector<double*> &y_double_gpu,
    size_t blocksize,
    size_t threadsperblock,
    int Niter
    ) 
{
    int nDevices;
    size_t dynsharedsize = 0;
    pu_gpuStream_t stream = 0;
    pu_gpuErrorCheck(pu_gpuGetDeviceCount(&nDevices));
    for (auto idev=0;idev<nDevices;idev++) {
        Log()<<" at device "<<idev<<" and doing stuff "<<std::endl;
        pu_gpuErrorCheck(pu_gpuSetDevice(idev));
        auto time_kernel = NewTimer();
        for (auto i=0; i<Niter;i++) {
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_int_gpu[idev], y_int_gpu[idev], N);
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_float_gpu[idev], y_float_gpu[idev], N);
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_double_gpu[idev], y_double_gpu[idev], N);
        }
        LogTimeTakenOnDevice(time_kernel);
    }
}

// __global__ void vector_square(const int *, int *, size_t);
// __global__ void vector_square(const float *, float *, size_t);
// __global__ void vector_square(const double *, double *, size_t);
