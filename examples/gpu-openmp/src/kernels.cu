#include "hip/hip_runtime.h"
#include "kernels.h"
#include "common.h"

template <typename T> __global__ 
void vector_square(const T *A_d, T *C_d, size_t N)
{
    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
}

void silly_template_instansiation()
{
    int ix,iy;
    float fx,fy;
    double dx,dy;
    size_t N;
    size_t blocksize = 256;
    size_t threadsperblock = 1024;
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &ix, &iy, N);
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &fx, &fy, N);
    pu_gpuLaunchKernel(vector_square, 
        dim3(blocksize), dim3(threadsperblock), 
        0, 0,
        &dx, &dy, N);
}

void compute_kernel1(size_t N, 
    std::vector<int*> &x_int_gpu, 
    std::vector<int*> &y_int_gpu, 
    std::vector<float*> &x_float_gpu, 
    std::vector<float*> &y_float_gpu, 
    std::vector<double*> &x_double_gpu, 
    std::vector<double*> &y_double_gpu,
    int Niter,
    size_t blocksize,
    size_t threadsperblock
    ) 
{
    int nDevices;
    size_t dynsharedsize = 0;
    pu_gpuStream_t stream = 0;
    pu_gpuErrorCheck(pu_gpuGetDeviceCount(&nDevices));
    Log()<<" Computing kernels ... "<<std::endl;
    for (auto idev=0;idev<nDevices;idev++) {
        pu_gpuErrorCheck(pu_gpuSetDevice(idev));
        auto time_kernel = NewTimer();
        for (auto i=0; i<Niter;i++) {
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_int_gpu[idev], y_int_gpu[idev], N);
#ifdef GPU_DEBUG
            pu_gpuCheckLastKernel();
#endif
            pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_float_gpu[idev], y_float_gpu[idev], N);
#ifdef GPU_DEBUG
                pu_gpuCheckLastKernel();
#endif
                pu_gpuLaunchKernel(vector_square, 
                dim3(blocksize), dim3(threadsperblock), 
                dynsharedsize, stream,
                x_double_gpu[idev], y_double_gpu[idev], N);
#ifdef GPU_DEBUG
                pu_gpuCheckLastKernel();
#endif
        }
        LogTimeTakenOnDevice(time_kernel);
    }
}

// __global__ void vector_square(const int *, int *, size_t);
// __global__ void vector_square(const float *, float *, size_t);
// __global__ void vector_square(const double *, double *, size_t);
